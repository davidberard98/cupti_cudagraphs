
#include <hip/hip_runtime.h>
#include <cstddef>
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include <sys/time.h>
#include <iostream>
#include <cupti.h>

#define N 500000 // tuned such that kernel takes a few microseconds

__global__ void shortKernel(float * out_d, float * in_d){
  int idx=blockIdx.x*blockDim.x+threadIdx.x;
  if(idx<N) out_d[idx]=1.23*in_d[idx];
}

#define CUPTI_CALL(call)                           \
  [&]() -> CUptiResult {                           \
    CUptiResult _status_ = call;                   \
    if (_status_ != CUPTI_SUCCESS) {               \
      const char* _errstr_ = nullptr;              \
      cuptiGetResultString(_status_, &_errstr_);   \
      std::cerr <<                  \
          "function " << #call << " failed with error " << _errstr_ << " (" << (int)_status_ << ")"; \
    }                                              \
    return _status_;                               \
  }()

#define CHECK(call)                                                      \
{                                                                        \
  const hipError_t error = call;                                        \
  if (error != hipSuccess) {                                            \
    printf("Error: %s:%d", __FILE__, __LINE__);                          \
    printf("code:%d, reason %s\n", error, hipGetErrorString(error));    \
    exit(1);                                                             \
  }                                                                      \
}

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec*1e-6);
}

struct TimerGuard {
  double start_time;
  std::string label_;
  TimerGuard(std::string label = "") : label_(label) {
    start_time = cpuSecond();
  }
  ~TimerGuard() {
    double end_time = cpuSecond();
    std::cout << " time " << label_ << ": " << int((end_time - start_time)*1e6) << " us " << std::endl;
  }
};

extern void initTrace(void);
extern void finiTrace(void);

void CUPTIAPI bufferRequestedTrampoline_(
  uint8_t** buffer,
  size_t* size,
	size_t* maxNumRecords) {
  // TODO
}

void CUPTIAPI bufferCompletedTrampoline_(
  hipCtx_t ctx,
  uint32_t streamId,
  uint8_t* buffer,
  size_t /* unused */,
	size_t validSize) {
  // TODO
}

void CUPTIAPI callback_switchboard_(
  CUpti_CallbackDomain domain,
  CUpti_CallbackId cbid,
  const CUpti_CallbackData* cbInfo) {
  // TODO
}
void myInitTrace(void) {
  CUpti_SubscriberHandle subscriber{0};
  CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_RUNTIME));
  CUPTI_CALL(cuptiActivityRegisterCallbacks(bufferRequestedTrampoline_, bufferCompletedTrampoline_));
  CUPTI_CALL(cuptiSubscribe(&subscriber, (CUpti_CallbackFunc)callback_switchboard_, NULL));
}

int main() {
#define NSTEP 1000
#define NKERNEL 20

  int threads = 512;
  int blocks = (N + threads - 1) / threads;

  hipStream_t stream;
  CHECK(hipStreamCreate(&stream));

  float *in_d, *out_d;
  CHECK(hipMalloc((float**)  &in_d, N*sizeof(float)));
  CHECK(hipMalloc((float**) &out_d, N*sizeof(float)));

  {
    TimerGuard guard("without cuda graph");
    for(int istep=0; istep<NSTEP; istep++){
      for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
        shortKernel<<<blocks, threads, 0, stream>>>(out_d, in_d);
        hipStreamSynchronize(stream);
      }
    }
  }

	bool graphCreated=false;
	hipGraph_t graph;
	hipGraphExec_t instance;

  myInitTrace();

  {
    TimerGuard guard("WITH cuda graph");
    for(int istep=0; istep<NSTEP; istep++){
      if(!graphCreated){
        hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
        for(int ikrnl=0; ikrnl<NKERNEL; ikrnl++){
          shortKernel<<<blocks, threads, 0, stream>>>(out_d, in_d);
        }
        hipStreamEndCapture(stream, &graph);
        hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
        graphCreated=true;
        hipGraphLaunch(instance, stream);
      }
      hipStreamSynchronize(stream);
    }
  }

  /*
  CUPTI_CALL(
      cuptiActivityRegisterCallbacks(bufferRequestedTrampoline, bufferCompletedTrampoline));

  CUPTI_CALL(cuptiActivityFlushAll(CUPTI_ACTIVITY_FLAG_FLUSH_FORCED));
  */

  return 0;
}
